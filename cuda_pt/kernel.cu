#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cmath>
#include "kernel.h"
#include "shape.h"
#include <thrust/random.h>

constexpr float PI = 3.141592653589793238462643383279502884197169399375105820974944f;

struct RenderInfo : RenderSettings {
    float pixelsize;
    Vec3 center_dir;
    Vec3 screeny;
    Vec3 screenx;


    DEVICE HOST RenderInfo(const RenderSettings &rs) : RenderSettings (rs) {
        pixelsize = rs.screenwidth / rs.WIDTH;
        center_dir = (rs.viewat - rs.viewfrom).normalized();
        screeny = (rs.upvec - center_dir * center_dir.dot(rs.upvec)).normalized();
        screenx = center_dir.cross(screeny);
    }
};

__device__ __host__ Vec3 uniformHemisphere(const Vec3 &nZ, thrust::ranlux24 &rgen)
{
    Vec3 x1 = nZ.cross(Vec3(1, 0, 0));
    Vec3 x2 = nZ.cross(Vec3(0, 1, 0));
    Vec3 nX = (x1.sqNorm() > x2.sqNorm()) ? (x1.normalized()) : (x2.normalized());
    Vec3 nY = nZ.cross(nX);

    thrust::uniform_real_distribution<float> dist_phi(-PI, PI);
    float phi = dist_phi(rgen);

    thrust::uniform_real_distribution<float> dist_costheta(0.0, 1.0);
    float costheta = dist_costheta(rgen);
    float theta = std::acos(costheta);

    return nX * (std::cos(phi) * std::sin(theta)) + nY * (std::sin(phi) * std::sin(theta)) + nZ * costheta;
}

__device__ __host__ Vec3 pixelproc(RenderInfo ri, size_t ixx, size_t ixy, const Object *objs, size_t objs_n, size_t npath, thrust::ranlux24 &rgen_g)
{
    //copy radom generator from global memory
    thrust::ranlux24 rgen = rgen_g;

    Sphere sphere({ 0, 10, 0 }, 5.0);
    thrust::uniform_real_distribution<float> dist(-0.5f, 0.5f);
    float dx = ri.pixelsize * (ixx - ri.WIDTH / 2.0f + dist(rgen)) ;
    float dy = ri.pixelsize * (ri.HEIGHT / 2.0f - ixy + dist(rgen));
    Vec3 pos = ri.viewat + dx * ri.screenx + dy * ri.screeny;

    Ray ray = Ray::FromTo(ri.viewfrom, pos);

    Vec3 ret = { 0, 0, 0 };
    Vec3 contrib = { 1, 1, 1 };
    for (int depth = 0; depth < 10; depth++) {
        bool intersect = false;
        size_t objid = 0;
        float nearest = 0;
        for (size_t i = 0; i < objs_n; i++) {
            float dist = objs[i].shape.firstIntersecttion(ray);
            if (dist > 0) {
                if (!intersect || dist < nearest) {
                    intersect = true;
                    objid = i;
                    nearest = dist;
                }
            }
        }

        if (intersect) {
            Vec3 hitpos = ray.atDistance(nearest);
            const auto &emission = objs[objid].optical.emission;
            const auto &reflection = objs[objid].optical.reflection;
            ret.x += emission.x * contrib.x;
            ret.y += emission.y * contrib.y;
            ret.z += emission.z * contrib.z;
            contrib.x *= reflection.x;
            contrib.y *= reflection.y;
            contrib.z *= reflection.z;


            Vec3 norm = objs[objid].shape.getNormal(hitpos);
            if (norm.dot(ray.n_dir) > 0) {
                norm = -1 * norm;
            }
            ray.n_dir = uniformHemisphere(norm, rgen);
            float direction_prob_density = 1 / (4 * PI);
            float brdf_coeff = 1 / (4 * PI);
            contrib = contrib * brdf_coeff / direction_prob_density;
            //ray.n_dir = ray.n_dir - 2 * norm * ray.n_dir.dot(norm);
            ray.start = hitpos + 1e-3f * ray.n_dir;

        }
        else {
            break;
        }
        if (contrib.norm() < 1e-6f) {
            break;
        }
    }
    //restore random generator to global memory
    rgen_g = rgen;
    return ret;
}

__global__ void initialize_rgens(RenderInfo ri, thrust::ranlux24 *rgens)
{
    size_t ixx = blockDim.x * blockIdx.x + threadIdx.x;
    size_t ixy = blockDim.y * blockIdx.y + threadIdx.y;

    size_t ix = ixy * ri.WIDTH + ixx;
    if (ixx < ri.WIDTH && ixy < ri.HEIGHT) {
        rgens[ix] = thrust::ranlux24(ix);
    }
}

__global__ void pixelproc_kernel(RenderInfo ri, Vec3 *map, Object *objs, size_t objs_n, size_t npath,thrust::ranlux24 *rgens)
{
    size_t ixx = blockDim.x * blockIdx.x + threadIdx.x;
    size_t ixy = blockDim.y * blockIdx.y + threadIdx.y;
    
    size_t ix = ixy * ri.WIDTH + ixx;
    if (ixx < ri.WIDTH && ixy < ri.HEIGHT) {
        //thrust::uniform_int_distribution<uint32_t> dist_seed;
        //seed random generator
        //don't use given generator to avoid passing a reference to the global memory
        //thrust::ranlux24 rgen(dist_seed(rgens[ix]));
        map[ix] = map[ix] + pixelproc(ri, ixx, ixy, objs, objs_n, npath, rgens[ix]);
    }
}



std::vector<Vec3> kernelmain(const RenderSettings &rs, const std::vector<Object> &objs, bool usegpu)
{
    RenderInfo ri(rs);
    printf("ex %f %f %f\n", ri.screenx.x, ri.screenx.y, ri.screenx.z);
    printf("ey %f %f %f\n", ri.screeny.x, ri.screeny.y, ri.screeny.z);
    printf("ez %f %f %f\n", -ri.center_dir.x, -ri.center_dir.y, -ri.center_dir.z);
    std::vector<Vec3> v(rs.HEIGHT * rs.WIDTH, { 0, 0, 0 });
    if (usegpu) {
        Vec3 *result;
        hipMalloc(&result, sizeof(Vec3) * rs.HEIGHT * rs.WIDTH);
        hipMemset(result, 0, sizeof(Vec3) * rs.HEIGHT * rs.WIDTH);

        Object *o;
        hipMalloc(&o, sizeof(Object) * objs.size());
        hipMemcpy(o, objs.data(), sizeof(Object) * objs.size(), hipMemcpyHostToDevice);

       
        dim3 threadsPerBlock(16, 16);
        dim3 blocks(rs.HEIGHT / 16 + 1, rs.WIDTH / 16 + 1);

        thrust::ranlux24 *rgens_dev;
        hipMalloc(&rgens_dev, sizeof(thrust::ranlux24) * ri.HEIGHT * ri.WIDTH);
        //seed random generators
        initialize_rgens <<<blocks, threadsPerBlock>>> (ri, rgens_dev);

        for (size_t i = 0; i < ri.PATH; i++) {
            if (i % 10 == 0) {
                printf("%zu / %zu\n", i, ri.PATH);
            }
            pixelproc_kernel<<<blocks, threadsPerBlock>>>(ri, result, o, objs.size(), i, rgens_dev);
            hipDeviceSynchronize();
        }
        hipMemcpy(v.data(), result, sizeof(Vec3) * rs.HEIGHT * rs.WIDTH, hipMemcpyDeviceToHost);
        hipFree(o);
        hipFree(result);
        hipFree(rgens_dev);
    }
    else {
      
        for (size_t ixy = 0; ixy < rs.HEIGHT; ixy++) {
            printf("%zu\n", ixy);
            for (size_t ixx = 0; ixx < rs.WIDTH; ixx++) {
                size_t ix = ixy * rs.WIDTH + ixx;
                thrust::ranlux24 rgen((uint32_t)(ixy * ri.HEIGHT + ixx));
                for (size_t i = 0; i < ri.PATH; i++) {
                    v.at(ix) = v.at(ix) + pixelproc(ri, ixx, ixy, objs.data(), objs.size(), i, rgen);
                }
            }
        }
    }
    for (auto &pixel : v) {
        pixel = pixel / (float)ri.PATH;
    }
    return v;
}